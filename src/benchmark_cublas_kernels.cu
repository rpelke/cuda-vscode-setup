#include "benchmark.cuh"
#include "utils.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

void free_sgemm_matrices(float *&d_A, float *&d_B, float *&d_C,
                         float *&d_C_init_helper) {
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_C_init_helper);
    d_A = nullptr;
    d_B = nullptr;
    d_C = nullptr;
    d_C_init_helper = nullptr;
}

int main() {
    int M = 1024;
    int N = 1024;
    int K = 1024;

    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr,
          *d_C_init_helper = nullptr;
    float alpha = 2.0f;
    float beta = 3.0f;

    Benchmark b;

    std::vector<float> h_A, h_B, h_C, res;
    b.init_matrices(h_A, h_B, h_C, res, M, N, K);
    b.copy_to_device(d_A, d_B, d_C, d_C_init_helper, h_A, h_B, h_C, res, M, N,
                     K);

    // Benchmark all gemmEx algorithms
    std::function<void(hipStream_t)> resetC = [=](hipStream_t stream) {
        CUDA_CHECK(hipMemcpyAsync(d_C, d_C_init_helper, M * N * sizeof(float),
                                   hipMemcpyDeviceToDevice, stream));
    };

    for (int i = HIPBLAS_GEMM_DEFAULT; i <= CUBLAS_GEMM_ALGO23; i++) {
        hipblasGemmAlgo_t algo = static_cast<hipblasGemmAlgo_t>(i);

        std::function<hipblasStatus_t(hipblasHandle_t)> kernel =
            [=](hipblasHandle_t handle) -> hipblasStatus_t {
            return hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
                                &alpha, d_B, HIP_R_32F, N, d_A, HIP_R_32F, K,
                                &beta, d_C, HIP_R_32F, N, HIP_R_32F, algo);
        };

        double cublas_ms = b.benchmark_cublas(kernel, resetC);
        std::cout << "SGEMM Algorithm " << i << " finished in " << cublas_ms
                  << " ms." << std::endl;
    }
    for (int i = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
         i <= CUBLAS_GEMM_ALGO15_TENSOR_OP; i++) {
        hipblasGemmAlgo_t algo = static_cast<hipblasGemmAlgo_t>(i);

        std::function<hipblasStatus_t(hipblasHandle_t)> kernel =
            [=](hipblasHandle_t handle) -> hipblasStatus_t {
            return hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K,
                                &alpha, d_B, HIP_R_32F, N, d_A, HIP_R_32F, K,
                                &beta, d_C, HIP_R_32F, N, HIP_R_32F, algo);
        };

        double cublas_ms = b.benchmark_cublas(kernel, resetC);
        std::cout << "SGEMM Algorithm " << i << " finished in " << cublas_ms
                  << " ms." << std::endl;
    }

    free_sgemm_matrices(d_A, d_B, d_C, d_C_init_helper);

    std::cout << "-------------------------" << std::endl;

    b.init_matrices(h_A, h_B, h_C, res, M, N);
    b.copy_to_device(d_A, d_B, d_C, d_C_init_helper, h_A, h_B, h_C, res, M, N);

    std::function<hipblasStatus_t(hipblasHandle_t)> kernel =
        [=](hipblasHandle_t handle) -> hipblasStatus_t {
        return hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, d_A, M, d_B, 1,
                              &beta, d_C, 1);
    };

    std::function<void(hipStream_t)> resetC2 = [=](hipStream_t stream) {
        CUDA_CHECK(hipMemcpyAsync(d_C, d_C_init_helper, M * sizeof(float),
                                   hipMemcpyDeviceToDevice, stream));
    };

    double cublas_ms = b.benchmark_cublas(kernel, resetC2);
    std::cout << "SGEMV finished in " << cublas_ms << " ms." << std::endl;

    free_sgemm_matrices(d_A, d_B, d_C, d_C_init_helper);
}
