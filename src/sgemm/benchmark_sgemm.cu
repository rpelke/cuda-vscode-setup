#include <chrono>
#include <cmath>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

#include "benchmark.cuh"
#include "sgemm/benchmark_sgemm.cuh"
#include "sgemm/sgemm.cuh"
#include "utils.cuh"

SGEMMBenchmark::SGEMMBenchmark() :
    d_A(nullptr), d_B(nullptr), d_C(nullptr), d_C_init_helper(nullptr) {}

double SGEMMBenchmark::benchmark_cpu(int M, int K, int N, float alpha,
                                     float beta) {
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_sgemm(M, N, K, alpha, h_A, h_B, beta, h_C_cpu);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    return std::chrono::duration<double, std::milli>(cpu_end - cpu_start)
        .count();
}

double SGEMMBenchmark::ms_to_gflops(int M, int K, int N, double ms) {
    double gflops = 2.0 * M * N * K / (ms * 1e6);
    return gflops;
}

void SGEMMBenchmark::benchmark_kernel(int M, int K, int N, float alpha,
                                      float beta, dim3 gridDim, dim3 blockDim,
                                      sgemm_kernel_t launcher,
                                      std::string kernel_name,
                                      float atol = 1e-2f) {
    copy_to_device(d_A, d_B, d_C, d_C_init_helper, h_A, h_B, h_C, h_C_init, M,
                   K, N);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    launcher(M, N, K, alpha, d_A, d_B, beta, d_C, gridDim, blockDim);
    hipEventRecord(stop);

    hipError_t err = hipEventSynchronize(stop);
    if (err != hipSuccess)
        printf("Kernel error: %s\n", hipGetErrorString(err));

    float kernel_ms;
    hipEventElapsedTime(&kernel_ms, start, stop);
    float kernel_gflops = ms_to_gflops(M, K, N, kernel_ms);
    copy_results_to_host(d_C, M, N, h_C);
    validate_results(h_C_cpu, h_C, kernel_name, M, N, atol);
    print_results(kernel_ms, kernel_gflops, kernel_name);
    free_device_mem(d_A, d_B, d_C, d_C_init_helper);
}

void SGEMMBenchmark::start_benchmarks(int M, int K, int N, float alpha,
                                      float beta) {
    // Initialize matrices
    init_matrices(h_A, h_B, h_C, h_C_init, h_C_cpu, h_C_cublas, M, K, N);

    // CPU reference
    double cpu_ms = benchmark_cpu(M, K, N, alpha, beta);
    double cpu_gflops = ms_to_gflops(M, K, N, cpu_ms);
    print_results(cpu_ms, cpu_gflops, "CPU");

    // Cublas reference
    copy_to_device(d_A, d_B, d_C, d_C_init_helper, h_A, h_B, h_C, h_C_init, M,
                   K, N);

    std::function<hipblasStatus_t(hipblasHandle_t)> func =
        [=](hipblasHandle_t handle) -> hipblasStatus_t {
        return hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha,
                            d_B, HIP_R_32F, N, d_A, HIP_R_32F, K, &beta, d_C,
                            HIP_R_32F, N, HIP_R_32F,
                            CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    };

    std::function<void(hipStream_t)> resetC = [=](hipStream_t stream) {
        CUDA_CHECK(hipMemcpyAsync(d_C, d_C_init_helper, M * N * sizeof(float),
                                   hipMemcpyDeviceToDevice, stream));
    };

    double cublas_ms = benchmark_cublas(func, resetC);

    double cublas_gflops = ms_to_gflops(M, K, N, cublas_ms);
    copy_results_to_host(d_C, M, N, h_C_cublas);
    validate_results(h_C_cpu, h_C_cublas, "Cublas", M, N, 5e-2f);
    print_results(cublas_ms, cublas_gflops, "Cublas");
    free_device_mem(d_A, d_B, d_C, d_C_init_helper);

    // 00: Test simple kernel
    dim3 blockDim_00(BLOCKSIZE_00, BLOCKSIZE_00, 1);
    dim3 gridDim_00(CEIL_DIV(M, BLOCKSIZE_00), CEIL_DIV(N, BLOCKSIZE_00), 1);
    benchmark_kernel(
        M, K, N, alpha, beta, gridDim_00, blockDim_00,
        [](int M, int K, int N, float alpha, const float *A, const float *B,
           float beta, float *C, dim3 gridDim, dim3 blockDim) -> void {
            sgemm_simple<<<gridDim, blockDim>>>(M, K, N, alpha, A, B, beta, C);
        },
        "Kernel 00");

    // 01: Test coalesced kernel
    dim3 blockDim_01(BLOCKSIZE_01 * BLOCKSIZE_01);
    dim3 gridDim_01(CEIL_DIV(N, BLOCKSIZE_01), CEIL_DIV(M, BLOCKSIZE_01), 1);
    benchmark_kernel(
        M, K, N, alpha, beta, gridDim_01, blockDim_01,
        [](int M, int K, int N, float alpha, const float *A, const float *B,
           float beta, float *C, dim3 gridDim, dim3 blockDim) -> void {
            sgemm_coalesced<<<gridDim, blockDim>>>(M, K, N, alpha, A, B, beta,
                                                   C);
        },
        "Kernel 01");

    // 02: Test tiled kernel
    dim3 blockDim_02(BLOCKSIZE_02, BLOCKSIZE_02, 1);
    dim3 gridDim_02(CEIL_DIV(N, BLOCKSIZE_02), CEIL_DIV(M, BLOCKSIZE_02), 1);
    benchmark_kernel(
        M, K, N, alpha, beta, gridDim_02, blockDim_02,
        [](int M, int K, int N, float alpha, const float *A, const float *B,
           float beta, float *C, dim3 gridDim, dim3 blockDim) -> void {
            sgemm_tiled<<<gridDim, blockDim>>>(M, K, N, alpha, A, B, beta, C);
        },
        "Kernel 02");

    // 03: Test tiled 2D kernel
    static_assert(BN_03 % TN_03 == 0 && BM_03 % TM_03 == 0,
                  "BN % TN != 0 || BM % TM != 0");
    static_assert(BN_03 / TN_03 == BK_03, "BN / TN != BK");
    static_assert(BM_03 / TM_03 == BK_03, "BM / TM != BK");
    static_assert(BK_03 >= TM_03 && BK_03 >= TN_03, "BK < TM || BK < TN");
    dim3 gridDim_03(CEIL_DIV(N, BN_03), CEIL_DIV(M, BM_03), 1);
    dim3 blockDim_03(BN_03 / TN_03, BM_03 / TM_03, 1);
    benchmark_kernel(
        M, K, N, alpha, beta, gridDim_03, blockDim_03,
        [](int M, int K, int N, float alpha, const float *A, const float *B,
           float beta, float *C, dim3 gridDim, dim3 blockDim) {
            sgemm_tiled_2d<<<gridDim, blockDim>>>(M, K, N, alpha, A, B, beta,
                                                  C);
        },
        "Kernel 03");

    // 04: Test tiled 2D kernel with vectorization
    static_assert(BN_04 % TN_04 == 0 && BM_04 % TM_04 == 0,
                  "BN % TN != 0 || BM % TM != 0");
    static_assert(BN_04 / TN_04 == BK_04, "BN / TN != BK");
    static_assert(BM_04 / TM_04 == BK_04, "BM / TM != BK");
    static_assert(BK_04 >= TM_04 && BK_04 >= TN_04, "BK < TM || BK < TN");
    static_assert(BK_04 >= VEC_SIZE_04 && BK_04 % VEC_SIZE_04 == 0,
                  "BK < VEC_SIZE_04 || BK % VEC_SIZE_04 != 0");
    dim3 gridDim_04(CEIL_DIV(N, BN_04), CEIL_DIV(M, BM_04), 1);
    dim3 blockDim_04(BN_04 / TN_04, BM_04 / TM_04, 1);
    benchmark_kernel(
        M, K, N, alpha, beta, gridDim_04, blockDim_04,
        [](int M, int K, int N, float alpha, const float *A, const float *B,
           float beta, float *C, dim3 gridDim, dim3 blockDim) {
            sgemm_tiled_2d_vectorized_1<<<gridDim, blockDim>>>(M, K, N, alpha,
                                                               A, B, beta, C);
        },
        "Kernel 04");

    // 05: Test tiled 2D kernel with vectorization
    static_assert(BN_05 % TN_05 == 0 && BM_05 % TM_05 == 0,
                  "BN % TN != 0 || BM % TM != 0");
    static_assert(BN_05 / TN_05 == BK_05, "BN / TN != BK");
    static_assert(BM_05 / TM_05 == BK_05, "BM / TM != BK");
    static_assert(BK_05 >= TM_05 && BK_05 >= TN_05, "BK < TM || BK < TN");
    static_assert(BK_05 >= VEC_SIZE_05 && BK_05 % VEC_SIZE_05 == 0,
                  "BK < VEC_SIZE_05 || BK % VEC_SIZE_05 != 0");
    static_assert((TN_05 >= VEC_SIZE_05) && (TN_05 % VEC_SIZE_05) == 0,
                  "TN < VEC_SIZE_05 || TN % VEC_SIZE_05 != 0");
    dim3 gridDim_05(CEIL_DIV(N, BN_05), CEIL_DIV(M, BM_05), 1);
    dim3 blockDim_05(BN_05 / TN_05, BM_05 / TM_05, 1);
    benchmark_kernel(
        M, K, N, alpha, beta, gridDim_05, blockDim_05,
        [](int M, int K, int N, float alpha, const float *A, const float *B,
           float beta, float *C, dim3 gridDim, dim3 blockDim) {
            sgemm_tiled_2d_vectorized_2<<<gridDim, blockDim>>>(M, K, N, alpha,
                                                               A, B, beta, C);
        },
        "Kernel 05");

    // 06: Test warptiling
    static_assert(BN_06 % WN_06 == 0 && BM_06 % WM_06 == 0,
                  "BN % WN != 0 || BM % WM != 0");
    static_assert(BN_06 / TN_06 == BK_06, "BN / TN != BK");
    static_assert(BM_06 / TM_06 == BK_06, "BM / TM != BK");
    static_assert(BK_06 >= TM_06 && BK_06 >= TN_06, "BK < TM || BK < TN");
    static_assert(WN_06 >= TN_06 && WM_06 >= TM_06, "WN < TN || WM < TM");
    static_assert(WN_06 % TN_06 == 0 && WM_06 % TM_06 == 0,
                  "WN % TN != 0 || WM % TM != 0");
    dim3 gridDim_06(CEIL_DIV(N, BN_06), CEIL_DIV(M, BM_06), 1);
    dim3 blockDim_06(BN_06 / TN_06, BM_06 / TM_06, 1);
    benchmark_kernel(
        M, K, N, alpha, beta, gridDim_06, blockDim_06,
        [](int M, int K, int N, float alpha, const float *A, const float *B,
           float beta, float *C, dim3 gridDim, dim3 blockDim) {
            sgemm_warptiling<<<gridDim, blockDim>>>(M, K, N, alpha, A, B, beta,
                                                    C);
        },
        "Kernel 06");

    // 07: Test tensor cores
    static_assert(TN_07 * TM_07 * 32 == BLOCKSIZE_07 * BLOCKSIZE_07);
    dim3 gridDim_07(CEIL_DIV(N, BLOCKSIZE_07), CEIL_DIV(M, BLOCKSIZE_07), 1);
    dim3 blockDim_07(BLOCKSIZE_07 / TN_07, BLOCKSIZE_07 / TM_07, 1);
    benchmark_kernel(
        M, K, N, alpha, beta, gridDim_07, blockDim_07,
        [](int M, int K, int N, float alpha, const float *A, const float *B,
           float beta, float *C, dim3 gridDim, dim3 blockDim) {
            sgemm_tensorcores<<<gridDim, blockDim>>>(M, K, N, alpha, A, B, beta,
                                                     C);
        },
        "Kernel 07", 1e-1f /*Higher tolerance due to fp32->fp16 conversion*/);
}
